#include "hip/hip_runtime.h"
//Christian Douglas
//Joseph Williamson
//Alex Trampert
//Riemann Summ Group Project
//Parallel code

#include <stdio.h>
#include "timer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define NUMBER_OF_THREADS 10240

// CUDA kernel
__global__ void findArea(float *n, float a, float b)
{	__shared__ float dX;
	dX = (b - a)/NUMBER_OF_THREADS;
    // Get our thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
	
    if (id == 0 ) {
        n[id] = (a * a) + (a * 3) + 3;
	}
	if (id == NUMBER_OF_THREADS-1){
		n[id] = (b * b) + (b * 3) + 3;
    } else if(id != 0) {
		float x = id * dX;
		n[id] = (2 * ((x * x) + (x * 3) + 3));
    }
}

int main()
{	
    //timer variables
    double start, finish, elapsed;
    //host variables
	float a = 0, b = 100;
	float sum = 0;
	const int size = NUMBER_OF_THREADS*sizeof(float);
    float n[size];
	
	float total = 0;
    //device variables
	float* n_dev;

    //allocate on gpu 
    hipMalloc((void**)&n_dev, size);
    hipMemcpy(n_dev, n, size, hipMemcpyHostToDevice);

    dim3 GridDim(10,1);
    dim3 BlockDim(1024,1);

    GET_TIME(start);
    //call kernel 
    findArea<<<GridDim,BlockDim>>>(n_dev, a, b);
    hipDeviceSynchronize();
    GET_TIME(finish);
    elapsed = finish - start;

	hipMemcpy(n, n_dev, size, hipMemcpyDeviceToHost);
    hipFree(n_dev);
    
	for (int i = 0; i < NUMBER_OF_THREADS; i++) 
	{	
        sum += n[i];
	}
    
	total = ((b - a)/(2 * NUMBER_OF_THREADS)) * sum;
	
    printf("Riemann's Sum: %f\n", total);
    printf("The code to be timed took %e seconds\n", elapsed);

    return 0;
}

